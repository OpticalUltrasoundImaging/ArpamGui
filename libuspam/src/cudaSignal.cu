#include "hip/hip_runtime.h"
/**
Everything here assumes column major for 2D arrays
*/
#include "uspam/cudaSignal.cuh"
#include "uspam/cudaUtil.cuh"

#include <array>
#include <complex>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <cuda/std/cmath>

__global__ void kernelHilbert_r2c_freq_switch(hipfftDoubleComplex *data,
                                              int rows, int cols) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x; // row
  const int j = blockIdx.y * blockDim.y + threadIdx.y; // col
  if (i < rows && i < cols) {
    const int idx = j * rows + i;
    // data[idx] *= -1j;
    data[idx] = hipCmul(data[idx], make_hipDoubleComplex(0, -1));
  }
}

__global__ void kernelHilbert_r2c_scale_and_abs(double *real, double *imag,
                                                int rows, int cols) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < rows && j < cols) {
    const int idx = j * rows + i;
    real[idx] = hipCabs(
        make_hipDoubleComplex(real[idx], imag[idx] / static_cast<double>(rows)));
  }
}

void hilbert2(const double *x, double *out, const int rows, const int cols) {
  hipfftHandle planr2c;
  hipfftHandle planc2r;
  hipStream_t stream = nullptr;

  std::array<int, 1> fft_size = {rows};
  const int dist = rows;
  const int stride = 1;

  CUFFT_CALL(hipfftCreate(&planr2c));
  CUFFT_CALL(hipfftCreate(&planc2r));
  // clang-format off
  CUFFT_CALL(hipfftPlanMany(&planr2c, fft_size.size(), fft_size.data(),
                           nullptr, stride, dist,
                           nullptr, stride, dist,
                           HIPFFT_D2Z, cols));

  CUFFT_CALL(hipfftPlanMany(&planc2r, fft_size.size(), fft_size.data(),
                           nullptr, stride, dist,
                           nullptr, stride, dist,
                           HIPFFT_Z2D, cols));
  // clang-format on
  CUDA_RT_CALL(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUFFT_CALL(hipfftSetStream(planr2c, stream));
  CUFFT_CALL(hipfftSetStream(planc2r, stream));

  // Create device arrays
  hipfftDoubleComplex *d_cx;
  double *d_real;
  double *d_imag;

  CUDA_RT_CALL(hipMalloc(&d_cx, rows * cols * sizeof(hipfftDoubleComplex)));
  CUDA_RT_CALL(hipMalloc(&d_real, rows * cols * sizeof(double)));
  CUDA_RT_CALL(hipMalloc(&d_imag, rows * cols * sizeof(double)));

  CUDA_RT_CALL(hipMemcpyAsync(d_real, x, rows * cols * sizeof(double),
                               hipMemcpyHostToDevice, stream));

  // Forward fft
  CUFFT_CALL(hipfftExecD2Z(planr2c, d_real, d_cx));

  // Manipulate spectrum for Hilbert transform
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((rows + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (cols + threadsPerBlock.y - 1) / threadsPerBlock.y);
  kernelHilbert_r2c_freq_switch<<<numBlocks, threadsPerBlock, 0, stream>>>(
      d_cx, rows, cols);

  // Backward fft
  CUFFT_CALL(hipfftExecZ2D(planc2r, d_cx, d_imag));

  kernelHilbert_r2c_scale_and_abs<<<numBlocks, threadsPerBlock, 0, stream>>>(
      d_real, d_imag, rows, cols);

  // Copy to output
  CUDA_RT_CALL(hipMemcpyAsync(out, d_real, rows * cols * sizeof(double),
                               hipMemcpyDeviceToHost, stream));

  CUDA_RT_CALL(hipStreamSynchronize(stream));

  // Cleanup
  CUDA_RT_CALL(hipFree(d_cx));
  CUDA_RT_CALL(hipFree(d_real));
  CUDA_RT_CALL(hipFree(d_imag));

  CUDA_RT_CALL(hipfftDestroy(planr2c));
  CUDA_RT_CALL(hipfftDestroy(planc2r));

  CUDA_RT_CALL(hipStreamDestroy(stream));
}

