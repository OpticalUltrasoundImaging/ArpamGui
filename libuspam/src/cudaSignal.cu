#include "hip/hip_runtime.h"
/**
Everything here assumes column major for 2D arrays
*/
// NOLINTBEGIN(*-pointer-arithmetic, *-trailing-return-type, *-const-cast)
#include <hip/hip_runtime.h>
#include <>

#include <hip/hip_complex.h>
#include <cuda/std/cmath>
#include <hipfft/hipfft.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#include "uspam/cudaSignal.h"
#include "uspam/cudaUtil.h"
#include <array>
#include <map>
#include <tuple>

__global__ void convolve1DSame(const double *in, const double *kernel,
                               double *output, int inSize, int batchSize,
                               int kernelSize) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;
  const int k_half = kernelSize / 2;
  double sum = 0;

  if (i < inSize && j < batchSize) {
    int offset = j * inSize;
    for (int k = 0; k < kernelSize; k++) {
      int idx = i + k_half - k; // Center the kernel on the current element
      if (idx >= 0 && idx < inSize) {
        sum += in[offset + idx] * kernel[k];
      }
    }
    output[offset + i] = sum;
  }
}

// FIR filter
void uspam::cuda::firFilt2_same_device(const double *in, const double *kernel,
                                       double *out, const int inSize,
                                       const int batchSize,
                                       const int kernelSize,
                                       hipStream_t stream) {
  dim3 threadsPerBlock(32, 16);
  dim3 numBlocks((inSize + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (batchSize + threadsPerBlock.y - 1) / threadsPerBlock.y);
  convolve1DSame<<<numBlocks, threadsPerBlock, 0, stream>>>(
      in, kernel, out, inSize, batchSize, kernelSize);
}

__global__ void kernelHilbert_r2c_freq_switch(hipfftDoubleComplex *data,
                                              const uint32_t rows,
                                              const uint32_t cols) {
  const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; // row
  const uint32_t j = blockIdx.y * blockDim.y + threadIdx.y; // col
  if (i < rows && i < cols) {
    const uint32_t idx = j * rows + i;
    // data[idx] *= -1j;
    data[idx] = hipCmul(data[idx], make_hipDoubleComplex(0, -1));
  }
}

__global__ void kernelHilbert_r2c_scale_and_abs(const double *real,
                                                const double *imag, double *out,
                                                const uint32_t rows,
                                                const uint32_t cols) {
  const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < rows && j < cols) {
    const uint32_t idx = j * rows + i;
    out[idx] = hipCabs(
        make_hipDoubleComplex(real[idx], imag[idx] / static_cast<double>(rows)));
  }
}

void uspam::cuda::hilbert2_ref(const double *in, double *out, const int fftSize,
                               const int batchSize, hipStream_t stream) {
  hipfftHandle planr2c{};
  hipfftHandle planc2r{};

  std::array<int, 1> _fftSize = {fftSize};
  const int dist = fftSize;
  const int stride = 1;

  CUFFT_CALL(hipfftCreate(&planr2c));
  CUFFT_CALL(hipfftCreate(&planc2r));
  // clang-format off
  CUFFT_CALL(hipfftPlanMany(&planr2c, _fftSize.size(), _fftSize.data(),
                           nullptr, stride, dist,
                           nullptr, stride, dist,
                           HIPFFT_D2Z, batchSize));

  CUFFT_CALL(hipfftPlanMany(&planc2r, _fftSize.size(), _fftSize.data(),
                           nullptr, stride, dist,
                           nullptr, stride, dist,
                           HIPFFT_Z2D, batchSize));
  // clang-format on
  CUFFT_CALL(hipfftSetStream(planr2c, stream));
  CUFFT_CALL(hipfftSetStream(planc2r, stream));

  // Create device arrays
  hipfftDoubleComplex *d_cx = nullptr;
  double *d_real = nullptr;
  double *d_imag = nullptr;

  CUDA_RT_CALL(
      hipMalloc(&d_cx, fftSize * batchSize * sizeof(hipfftDoubleComplex)));
  CUDA_RT_CALL(hipMalloc(&d_real, fftSize * batchSize * sizeof(double)));
  CUDA_RT_CALL(hipMalloc(&d_imag, fftSize * batchSize * sizeof(double)));

  CUDA_RT_CALL(hipMemcpyAsync(d_real, in, fftSize * batchSize * sizeof(double),
                               hipMemcpyHostToDevice, stream));

  // Forward fft
  CUFFT_CALL(hipfftExecD2Z(planr2c, d_real, d_cx));

  // Manipulate spectrum for Hilbert transform
  dim3 threadsPerBlock(32, 16);
  dim3 numBlocks((fftSize + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (batchSize + threadsPerBlock.y - 1) / threadsPerBlock.y);
  kernelHilbert_r2c_freq_switch<<<numBlocks, threadsPerBlock, 0, stream>>>(
      d_cx, fftSize, batchSize);

  // Backward fft
  CUFFT_CALL(hipfftExecZ2D(planc2r, d_cx, d_imag));

  kernelHilbert_r2c_scale_and_abs<<<numBlocks, threadsPerBlock, 0, stream>>>(
      d_real, d_imag, d_real, fftSize, batchSize);

  // Copy to output
  CUDA_RT_CALL(hipMemcpyAsync(out, d_real,
                               fftSize * batchSize * sizeof(double),
                               hipMemcpyDeviceToHost, stream));

  CUDA_RT_CALL(hipStreamSynchronize(stream));

  // Cleanup
  CUDA_RT_CALL(hipFree(d_cx));
  CUDA_RT_CALL(hipFree(d_real));
  CUDA_RT_CALL(hipFree(d_imag));

  CUDA_RT_CALL(hipfftDestroy(planr2c));
  CUDA_RT_CALL(hipfftDestroy(planc2r));
}

// fft_size, batch_size
using HilbertPlanKey = std::tuple<int, int>;
struct HilbertPlan {
  // Plans
  hipfftHandle planr2c{};
  hipfftHandle planc2r{};

  // Device arrays
  hipfftDoubleComplex *d_cx{};
  double *d_real{};
  double *d_imag{};

  explicit HilbertPlan(const HilbertPlanKey &key) {
    const auto [_fftSize, batch_size] = key;

    std::array<int, 1> fftSize = {_fftSize};
    const int dist = _fftSize;
    const int stride = 1;

    CUFFT_CALL(hipfftCreate(&planr2c));
    CUFFT_CALL(hipfftCreate(&planc2r));

    // clang-format off
    CUFFT_CALL(hipfftPlanMany(&planr2c, fftSize.size(), fftSize.data(),
                             nullptr, stride, dist,
                             nullptr, stride, dist,
                             HIPFFT_D2Z, batch_size));
    CUFFT_CALL(hipfftPlanMany(&planc2r, fftSize.size(), fftSize.data(),
                             nullptr, stride, dist,
                             nullptr, stride, dist,
                             HIPFFT_Z2D, batch_size));
    // clang-format on

    const auto N = _fftSize * batch_size;
    CUDA_RT_CALL(hipMalloc(&d_cx, N * sizeof(hipfftDoubleComplex)));
    CUDA_RT_CALL(hipMalloc(&d_real, N * sizeof(double)));
    CUDA_RT_CALL(hipMalloc(&d_imag, N * sizeof(double)));
  }
  HilbertPlan(HilbertPlan &) = delete;
  HilbertPlan(HilbertPlan &&) = delete;
  HilbertPlan &operator=(HilbertPlan &) = delete;
  HilbertPlan &operator=(HilbertPlan &&) = delete;

  ~HilbertPlan() {
    CUDA_RT_CALL(hipFree(d_cx));
    CUDA_RT_CALL(hipFree(d_real));
    CUDA_RT_CALL(hipFree(d_imag));

    CUDA_RT_CALL(hipfftDestroy(planr2c));
    CUDA_RT_CALL(hipfftDestroy(planc2r));
  }

  void setStream(hipStream_t stream) const {
    CUFFT_CALL(hipfftSetStream(planr2c, stream));
    CUFFT_CALL(hipfftSetStream(planc2r, stream));
  }
};

template <class Key, class Val> auto get_cached(const Key &key) {
  static thread_local std::map<Key, std::unique_ptr<Val>> cache;
  auto &val = cache[key];
  if (val == nullptr) {
    val = std::make_unique<Val>(key);
  }
  return val.get();
}

void uspam::cuda::hilbert2(const double *in, double *out, const int fftSize,
                           const int batchSize, hipStream_t stream) {
  auto &plan = *get_cached<HilbertPlanKey, HilbertPlan>({fftSize, batchSize});
  plan.setStream(stream);

  CUDA_RT_CALL(hipMemcpyAsync(plan.d_real, in,
                               fftSize * batchSize * sizeof(double),
                               hipMemcpyHostToDevice, stream));

  // Forward fft
  CUFFT_CALL(hipfftExecD2Z(plan.planr2c, plan.d_real, plan.d_cx));

  // Manipulate spectrum for Hilbert transform
  dim3 threadsPerBlock(32, 16);
  dim3 numBlocks((fftSize + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (batchSize + threadsPerBlock.y - 1) / threadsPerBlock.y);
  kernelHilbert_r2c_freq_switch<<<numBlocks, threadsPerBlock, 0, stream>>>(
      plan.d_cx, fftSize, batchSize);

  // Backward fft
  CUFFT_CALL(hipfftExecZ2D(plan.planc2r, plan.d_cx, plan.d_imag));

  kernelHilbert_r2c_scale_and_abs<<<numBlocks, threadsPerBlock, 0, stream>>>(
      plan.d_real, plan.d_imag, plan.d_real, fftSize, batchSize);

  // Copy to output
  CUDA_RT_CALL(hipMemcpyAsync(out, plan.d_real,
                               fftSize * batchSize * sizeof(double),
                               hipMemcpyDeviceToHost, stream));
}

void uspam::cuda::hilbert2_device(const double *device_in, double *device_out,
                                  const int fftSize, const int batchSize,
                                  hipStream_t stream) {
  auto &plan = *get_cached<HilbertPlanKey, HilbertPlan>({fftSize, batchSize});
  plan.setStream(stream);

  // Forward fft
  CUFFT_CALL(
      hipfftExecD2Z(plan.planr2c, const_cast<double *>(device_in), plan.d_cx));

  // Manipulate spectrum for Hilbert transform
  dim3 threadsPerBlock(32, 16);
  dim3 numBlocks((fftSize + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (batchSize + threadsPerBlock.y - 1) / threadsPerBlock.y);
  kernelHilbert_r2c_freq_switch<<<numBlocks, threadsPerBlock, 0, stream>>>(
      plan.d_cx, fftSize, batchSize);

  // Backward fft
  CUFFT_CALL(hipfftExecZ2D(plan.planc2r, plan.d_cx, plan.d_imag));

  kernelHilbert_r2c_scale_and_abs<<<numBlocks, threadsPerBlock, 0, stream>>>(
      device_in, plan.d_imag, device_out, fftSize, batchSize);
}

double calcDynamicRange_device(const double *d_in, int size, double noiseFloor,
                               hipStream_t stream) {
  thrust::device_ptr<const double> in(d_in);

  const auto peakIter =
      thrust::max_element(thrust::cuda::par.on(stream), in, in + size);
  const double peakLevel = *peakIter;
  const double dynamicRangeDB = 20.0 * std::log10(peakLevel / noiseFloor);

  return dynamicRangeDB;
}

void uspam::cuda::logCompress_device(const double *d_in, double *d_out,
                                     int size, double noiseFloor,
                                     double desiredDynamicRangeDB,
                                     hipStream_t stream) {
  thrust::device_ptr<const double> in(d_in);
  thrust::device_ptr<double> out(d_out);

  // Apply log compression with clipping
  thrust::transform(thrust::cuda::par.on(stream), in, in + size, out,
                    [=] __device__(const double val) {
                      double normVal = val / noiseFloor;
                      double compVal = (normVal > 0 ? 20 * log10(normVal) : 0);
                      compVal = max(compVal, 0.0);
                      compVal = min(compVal, desiredDynamicRangeDB);
                      return compVal / desiredDynamicRangeDB;
                    });
}

void uspam::cuda::logCompress_device(const thrust::device_vector<double> &d_in,
                                     thrust::device_vector<double> &d_out,
                                     double noiseFloor,
                                     double desiredDynamicRangeDB,
                                     hipStream_t stream) {

  d_out.resize(d_in.size());
  logCompress_device(thrust::raw_pointer_cast(d_in.data()),
                     thrust::raw_pointer_cast(d_out.data()), d_in.size(),
                     noiseFloor, desiredDynamicRangeDB, stream);
}

// __global__ void kernelLogCompress(double *d_in, double *d_out, int size,
//                                   double noiseFloor,
//                                   double desiredDynamicRangeDB) {
//   const int i = blockIdx.x * blockDim.x + threadIdx.x;

//   if (i < size) {
//     const double val = d_in[i];
//     double compressed = 20.0 * log10(val / noiseFloor);
//     compressed = max(compressed, double(0));
//     compressed = min(compressed, desiredDynamicRangeDB);
//     d_out[i] = compressed / desiredDynamicRangeDB;
//   }
// }

// double uspam::cuda::logCompress_device(double *_in, double *_out, int size,
//                                        double noiseFloor,
//                                        double desiredDynamicRangeDB,
//                                        hipStream_t stream) {
//   // hipStream_t streamMax;
//   // CUDA_RT_CALL(hipStreamCreate(&streamMax));

//   // const double peakLevel =
//   //     *thrust::max_element(thrust::cuda::par.on(streamMax), _in, _in +
//  size);
//  // const double dynamicRangeDB = 20.0 * std::log10(peakLevel / noiseFloor);
//  const double dynamicRangeDB = 0.;

//  int blockSize = 512;
//  int numBlocks = (size + blockSize - 1) / blockSize;
//  kernelLogCompress<<<numBlocks, blockSize, 0, stream>>>(
//      _in, _out, size, noiseFloor, desiredDynamicRangeDB);

//  // CUDA_RT_CALL(hipGetLastError());
//  // CUDA_RT_CALL(hipStreamDestroy(streamMax));

//  return dynamicRangeDB;
// }

// NOLINTEND(*-pointer-arithmetic, *-trailing-return-type, *-const-cast)